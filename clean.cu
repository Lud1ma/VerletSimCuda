#include "hip/hip_runtime.h"
#define WINDOW_X 1000
#define WINDOW_Y 800

#define PARTICLES 5000
#define CIRCLE_EDGES 64
#define SUB_STEPS 8
#define GRAVITY_X 0.f
#define GRAVITY_Y .5f
#define FRICTION .999f
#define BOUNCE .9f
#define MARGIN 0.0f
#define HEAT_COEFFICENT 0.1

#include <iostream>
#include <boost/random.hpp>
#include <math.h>

#include <SFML/Graphics.hpp>

__global__ void move(float *position_x, float *position_y, float *old_position_x, float *old_position_y, float *radius, int N) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    if (tid > N) {
        return;
    }
    
    float d_x = position_x[tid] - old_position_x[tid];
    float d_y = position_y[tid] - old_position_y[tid];
    
    old_position_x[tid] = position_x[tid];
    old_position_y[tid] = position_y[tid];
    
    position_x[tid] = position_x[tid] + (d_x + GRAVITY_X) * FRICTION;
    position_y[tid] = position_y[tid] + (d_y + GRAVITY_Y) * FRICTION;
}

__global__ void boundaries(float *position_x, float *position_y, float *old_position_x, float *old_position_y, float *radius, int N) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    if (tid > N) {
        return;
    }
    
    float d_x = position_x[tid] - old_position_x[tid];
    float d_y = position_y[tid] - old_position_y[tid];
    
    if (position_x[tid] - radius[tid] < 0) {
        position_x[tid] = radius[tid];
        old_position_x[tid] = position_x[tid] +  d_x * BOUNCE;
    } else if (position_x[tid] + radius[tid] > WINDOW_X) {
        position_x[tid] = WINDOW_X - radius[tid];
        old_position_x[tid] = position_x[tid] +  d_x * BOUNCE;
    }
    
    if (position_y[tid] - radius[tid] < 0) {
        position_y[tid] = radius[tid];
        old_position_y[tid] = position_y[tid] +  d_y * BOUNCE;
    } else if (position_y[tid] + radius[tid] > WINDOW_Y) {
        position_y[tid] = WINDOW_Y - radius[tid];
        old_position_y[tid] = position_y[tid] +  d_y * BOUNCE;
    }
}

__global__ void collision(float *position_x, float *position_y, float *old_position_x, float *old_position_y, float *radius, int N) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
 
    if (tid > N) {
        return;
    }
    
    for (int i = 0; i < N; i++) {
        if (tid != i) {
            float dir_x = position_x[tid] - position_x[i];
            float dir_y = position_y[tid] - position_y[i];
            float min = radius[i] + radius[tid];
            float dist = sqrt(dir_x * dir_x + dir_y * dir_y);
            
            if (dist <= min) {
                position_x[tid] = position_x[tid] + ((dir_x / dist) * ((min-dist) / 2));
                position_y[tid] = position_y[tid] + ((dir_y / dist) * ((min-dist) / 2));
                position_x[i] = position_x[i] - ((dir_x / dist) * ((min-dist) / 2));
                position_y[i] = position_y[i] - ((dir_y / dist) * ((min-dist) / 2));
                
//                 position_x[tid] = 0;
//                 position_x[i] = 0;
            }
        }
        
//         p1->setPosition(p1->getPosition()-((direction/act_distance)*((min_distance-act_distance)/2)));
//                     p2->setPosition(p2->getPosition()+((direction/act_distance)*((min_distance-act_distance)/2)));
    }
}

float *position_x, *position_y, *old_position_x, *old_position_y, *radius;
int ACTIVE_PARTICLES = 0;

void spawn(float pos_x, float pos_y, float old_pos_x, float old_pos_y, float rad) {
    position_x[ACTIVE_PARTICLES] = pos_x;
    position_y[ACTIVE_PARTICLES] = pos_y;   
    old_position_x[ACTIVE_PARTICLES] = old_pos_x;
    old_position_y[ACTIVE_PARTICLES] = old_pos_y;
    radius[ACTIVE_PARTICLES] = rad;
    ACTIVE_PARTICLES++;
}

int main() {
    size_t bytes = sizeof(float) * PARTICLES;

    
    hipMallocManaged(&position_x, bytes);
    hipMallocManaged(&position_y, bytes);
    hipMallocManaged(&old_position_x, bytes);
    hipMallocManaged(&old_position_y, bytes);
    hipMallocManaged(&radius, bytes);
    
    int laps = 0;
    
    // NUMTHREADS
    int BLOCK_SIZE = 1 << 10;
    int GRID_SIZE = (PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // Mainloop
    sf::RenderWindow* window = new sf::RenderWindow(sf::VideoMode(WINDOW_X, WINDOW_Y), "Verlet Simulator CUDA");
    sf::CircleShape circle = sf::CircleShape(1., CIRCLE_EDGES); // Circle
    
    //window->setFramerateLimit(60);
    while (window->isOpen()) {
        sf::Event event;
        while (window->pollEvent(event)) {
            if (event.type == sf::Event::Closed) {
                window->close();
            }
        }
        
        window->clear(sf::Color::Black);
        
        float x, y;
        
        if (ACTIVE_PARTICLES < PARTICLES) {
            if (ACTIVE_PARTICLES < 500) {
                spawn(50.f, 700.f, 40.f, 700.f, rand() % 5);
            } else {
                if (laps % 2 == 0) {
                    x = rand() % WINDOW_X;
                    y = rand() % 400;
                    spawn(x, y, x, y, rand() % 5);
                }
            }
        } 
        
        for (int i = 0; i < PARTICLES; i++) {
            circle.setPosition(sf::Vector2f(position_x[i], position_y[i])-sf::Vector2f(radius[i], radius[i]));
            circle.setFillColor(sf::Color::Red);
            circle.setRadius(radius[i]);
            window->draw(circle);
        }
        
        move<<<GRID_SIZE, BLOCK_SIZE>>>(position_x, position_y, old_position_x, old_position_y, radius, ACTIVE_PARTICLES);
        hipDeviceSynchronize();
        
        for (int i = 0; i < SUB_STEPS; i++) {
            boundaries<<<GRID_SIZE, BLOCK_SIZE>>>(position_x, position_y, old_position_x, old_position_y, radius, ACTIVE_PARTICLES);
            hipDeviceSynchronize(); 
            collision<<<GRID_SIZE, BLOCK_SIZE>>>(position_x, position_y, old_position_x, old_position_y, radius, ACTIVE_PARTICLES);
            hipDeviceSynchronize();
            //std::cout << position_x[1] - position_x[0] << "    " << position_y[1] - position_y[0] << "    " << radius[0] + radius[1]  << "    " << sqrt((position_x[1] - position_x[0]) * (position_x[1] - position_x[0]) + (position_y[1] - position_y[0]) * (position_y[1] - position_y[0])) << "    " << (sqrt((position_x[1] - position_x[0]) * (position_x[1] - position_x[0]) + (position_y[1] - position_y[0]) * (position_y[1] - position_y[0])) < radius[0] + radius[1]) << std::endl;;
        }
        
        window->display();
        laps++;
        std::cout << ACTIVE_PARTICLES << std::endl;
    }
    
    hipFree(position_x);
    hipFree(position_y);
    hipFree(old_position_x);
    hipFree(old_position_y);
    hipFree(radius);
}
